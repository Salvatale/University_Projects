
#include <hip/hip_runtime.h>
#include <iostream> 
#include <stdio.h>
#include <math.h>

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


const int threadsPerBlock=256;

__global__ void add( float *res ) {
	   __shared__ float shr[threadsPerBlock];

        

	   //// qui ogni thread scrive il proprio indice
	   //// da modificare con il calcolo del proprio relativo pezzo di rettangoli
	   //// utilizzare blockIdx.x, blockDim.x e gridDim.x per calcolare la propria posizione e le divisioni da gestire

       //id del thread
       const int tid = threadIdx.x + blockIdx.x * blockDim.x;

       float h = 1.0 /(blockDim.x * gridDim.x);

       float x = h*(tid - 0.5);

       //Utilizzo la f1(x) = (1.0 / (1.0 + x*x)) 
	double f1 = 1.0 / (1.0 + x*x);
	// f2(x) = sqrt(1-x*x);
	// double f2 = sqrt(1-x*x);
	shr[threadIdx.x] = 4*h*f1;

	   __syncthreads();
// for reductions, threadsPerBlock must be a power of 2 // because of the following code
   int i = blockDim.x/2;
   while (i != 0) {
   	 if (threadIdx.x < i)
	 shr[threadIdx.x] += shr[threadIdx.x + i];
	 __syncthreads();
	 i /= 2;
   }
   if (threadIdx.x==0)
    res[blockIdx.x] = shr[threadIdx.x];
}

int main(int argc,char** argv ) { 
    
    double  PI = 3.14159265358979323846264338327950288 ;

    int nblocks=128;
    //printf("hi\n");

    //In questo modo posso impostare il numero di blocchi che voglio da linea di comando
    if(argc == 2){
	nblocks = atoi(argv[1]);
    }
    // Creazione timer	    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);



    float* res=(float*)malloc(nblocks*sizeof(float));
    float *dev_res;
    HANDLE_ERROR( hipMalloc( (void**)&dev_res, nblocks*sizeof(float) ) );
//printf("start\n");
    
    hipEventRecord(start);
    add<<<nblocks,threadsPerBlock>>>( dev_res );
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float time = 0;
    hipEventElapsedTime(&time, start, stop);
   
    HANDLE_ERROR( hipMemcpy( res, dev_res, nblocks*sizeof(float), hipMemcpyDeviceToHost ) ); 
//printf("ok %f\n",res[0]);

    float total=0;
    for (int i=0;i<nblocks;i++){
     // printf("Block %d: %f\n",i,res[i]);
      total+=res[i];
    }

   //printf("Somma %f\n",total);
   hipFree( dev_res );
   fprintf(stderr,"%d,%1f,%1f,%1f\n",nblocks,time,fabs(PI-total),total);

    return 0; 
}
