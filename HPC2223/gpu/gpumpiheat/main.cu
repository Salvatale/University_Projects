#include "hip/hip_runtime.h"
#include "mpi.h"
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h> 

#define BLOCK_SIZE_X 16
#define BLOCK_SIZE_Y 16


__host__ __device__ int iDivUp(int a, int b){ return ((a % b) != 0) ? (a / b + 1) : (a / b); }

/***********************************/
/* JACOBI ITERATION FUNCTION - GPU */
/***********************************/
__global__ void Jacobi_Iterator_GPU(const float * __restrict__ T_old, float * __restrict__ T_new, const int NX, const int NY, float* recv, int rank)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x ;
    const int j = blockIdx.y * blockDim.y + threadIdx.y ;

    int left=i-1;
    int right=i+1;
    int up=j+1;
    int down=j-1;
    if (left<0) left=0;
    if (right>=gridDim.x*blockDim.x) right--;
    if (down<0) down=0;
    if (up>=gridDim.y*blockDim.y) up--; 
                              //                         N 
    int P = i + j*NX;         // node (i,j)              |
    int N = i + up*NX;        // node (i,j+1)            |
    int S = i + down*NX;      // node (i,j-1)     W ---- P ---- E
    int E = right + j*NX;     // node (i+1,j)            |
    int W = left + j*NX;      // node (i-1,j)            |
                              //                         S 
    float upv;
    if (rank==0 && j==NY-1) /// bordo alto -> N viene letto da recv
      upv=recv[i];
    else
      upv=T_old[N];
    float downv;
    if (rank==1 && j==0) /// bordo basso -> S viene letto da recv
      downv=recv[i];
    else
      downv=T_old[S];

    /// Update
    T_new[P] = 0.25 * (T_old[E] + T_old[W] + upv + downv);
}
/*************
***** copy constant
************/

__global__ void copy_constant(float * __restrict__ T, const float * __restrict__ T_const, const int NX, const int NY)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x ;
    const int j = blockIdx.y * blockDim.y + threadIdx.y ;

    int P = i + j*NX;

    /// Update
    float temp=T_const[P];
    if (temp>0)
       T[P] = temp;
}


//// questo kernel funziona con 1 blocco <=1024 thread
//// adattarlo per funzionare con piu' blocchi (es. NX=2048)
__global__ void copy_send(int base,float* d_t,float* d_send){

  /// P e' l'indice lineare sulla riga della matrice
  /// per ricavare la posizione, in caso di piu' blocchi,
  /// e' necessario costruire la linearizzazione come nei kernel sopra
  /// P = blockIdx.x*blockDim.x + threadIdx.x


  int P= threadIdx.x + base;

  d_send[threadIdx.x]=d_t[P];
}


/******************************/
/* TEMPERATURE INITIALIZATION */
/******************************/
void Initialize(float * __restrict h_T, const int NX, const int NY, int rank)
{
   int startx,starty,endx,endy;
	if (rank==0){
    startx=NX/2-NX/10;
    endx=NX/2+NX/10; 
    starty=NY/4;
    endy=NY-2; 
    }
    else{
    startx=NX/4-NX/10;
    endx=NX/4+NX/10; 
    starty=2;
    endy=NY-NY/4;     
    }
    for(int i=startx; i<endx; i++) 
        for(int j=starty; j<endy; j++) 
              h_T[i+j * NX] = 1.0;

}


void go(int rank, int rank_node){


    /// assuming two ranks
    /// y doubled, bottom part (y=0..NY-1) -> rank 0, top part (y=NY..2*NY-1) -> rank 1
 
    const int NX = 1024;         // --- Number of discretization points along the x axis
    const int NY = 1024;         // --- Number of discretization points along the y axis

    const int MAX_ITER = 100000;     // --- Number of Jacobi iterations

    // --- CPU temperature distributions
    float *h_T              = (float *)calloc(NX * NY, sizeof(float));
    Initialize(h_T,     NX, NY, rank);
    float *h_T_GPU_result   = (float *)malloc(NX * NY * sizeof(float));

    /// exchange buffer
    float *h_send   = (float *)malloc(NX * sizeof(float));
    float *h_recv   = (float *)malloc(NX * sizeof(float));

    // --- GPU temperature distribution
    float *d_T;     
    hipMalloc((void**)&d_T,      NX * NY * sizeof(float));
    float *d_T_old; 
    hipMalloc((void**)&d_T_old,  NX * NY * sizeof(float));
    float *d_T_const;     
    hipMalloc((void**)&d_T_const,  NX * NY * sizeof(float));

    /// exchange buffer on gpu
    float *d_send;     
    hipMalloc((void**)&d_send,  NX * sizeof(float));
    float *d_recv;     
    hipMalloc((void**)&d_recv,  NX * sizeof(float));

    hipMemcpy(d_T,     h_T, NX * NY * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_T_old, d_T, NX * NY * sizeof(float), hipMemcpyDeviceToDevice);
    hipMemcpy(d_T_const, d_T, NX * NY * sizeof(float), hipMemcpyDeviceToDevice);

    // --- Grid size
    dim3 dimBlock(BLOCK_SIZE_X, BLOCK_SIZE_Y);
    dim3 dimGrid (iDivUp(NX, BLOCK_SIZE_X), iDivUp(NY, BLOCK_SIZE_Y));

    double average_mpi=0;

    // --- Jacobi iterations on the device
    for (int k=0; k<MAX_ITER; k++) {
        copy_constant<<<dimGrid, dimBlock>>>(d_T,     d_T_const, NX, NY);   // --- Update d_T with constant data >0 stored in d_T_const

	// copy from matrix to send buffer (NX<=1024)
	int base; /// prepara la posizione iniziale 
	    	  /// nella matrice della riga da copiare su ram per la spedizione
	if (rank==0) base=(NY-1)*NX;
	else base=NY;

	/// se NX>1024 non ci sono abbastanza thread nel blocco!
        /// sostituire 1 con il calcolo simile a dimGrid per gestire NX thread a gruppi di 1024 thread
        /// l'equivalente di dimBlock diventa 1024 se NX>=1024 oppure NX se NX<1024
	copy_send<<<1,NX>>>(base,d_T,d_send);
	 
	// gpu -> cpu
	hipMemcpy(h_send,d_send,NX*sizeof(float),hipMemcpyDeviceToHost);

        double a=MPI_Wtime();
	MPI_Sendrecv(h_send,NX,MPI_FLOAT,1-rank,0,
		     h_recv,NX,MPI_FLOAT,1-rank,0,
		     MPI_COMM_WORLD,MPI_STATUS_IGNORE);
	double b=MPI_Wtime();
        average_mpi+=b-a;
	
	/// cpu -> gpu
	hipMemcpy(d_recv,h_recv,NX*sizeof(float),hipMemcpyHostToDevice);

        Jacobi_Iterator_GPU<<<dimGrid, dimBlock>>>(d_T,     d_T_old, NX, NY,d_recv,rank);   // --- Update d_T_old     starting from data stored in d_T
	float* temp;
	temp=d_T;
	d_T=d_T_old;
	d_T_old=temp;
    }

    // --- Copy result from device to host
    hipMemcpy(h_T_GPU_result, d_T, NX * NY * sizeof(float), hipMemcpyDeviceToHost);


  char name[256];
  sprintf(name,"test-%d.txt",rank);
  FILE* f=fopen(name,"w+");
    for (int j=0; j<NY; j++){
        for (int i=0; i<NX; i++) {
	    fprintf(f,"%2.2f ",h_T_GPU_result[j * NX + i]);
        }
	fprintf(f,"\n");
	}
  fclose(f);

    printf("Rank %d: average MPI communication time: %f mS\n",rank,1000*average_mpi/MAX_ITER);

    // --- Release host memory 
    free(h_T);
    free(h_T_GPU_result);

    // --- Release device memory
    hipFree(d_T);
    hipFree(d_T_old);









}

int stringCmp( const void *a, const void *b)
{
  return strcmp((char*)a,(char*)b);
}

 int main(int argc, char *argv[])
 {
 
 MPI_Init(&argc,&argv);

 char     host_name[MPI_MAX_PROCESSOR_NAME];
 char (*host_names)[MPI_MAX_PROCESSOR_NAME];
 MPI_Comm nodeComm;
 int  n, namelen, color, rank, nprocs;
 int rank_node, gpu_per_node; /// sul singolo nodo
 size_t bytes;
 int dev;
 struct hipDeviceProp_t deviceProp;

 MPI_Comm_rank(MPI_COMM_WORLD, &rank);
 MPI_Comm_size(MPI_COMM_WORLD, &nprocs);
 MPI_Get_processor_name(host_name,&namelen);

 bytes = nprocs * sizeof(char[MPI_MAX_PROCESSOR_NAME]);
 host_names = (char (*)[MPI_MAX_PROCESSOR_NAME]) malloc(bytes);

 strcpy(host_names[rank], host_name);
 for (n=0; n<nprocs; n++){
   MPI_Bcast(&(host_names[n]),MPI_MAX_PROCESSOR_NAME, MPI_CHAR, n, MPI_COMM_WORLD);
 }

 qsort(host_names, nprocs,  sizeof(char[MPI_MAX_PROCESSOR_NAME]), stringCmp);

 color = 0; /// linearizzazione dei processi su nodo uguale

 for (n=0; n<nprocs; n++){
   if(n>0&&strcmp(host_names[n-1], host_names[n])) color++;
   if(strcmp(host_name, host_names[n]) == 0) break;
 }

 if (rank==0){
   printf("Elenco ordinato rank -> host\n");
   
   for (n=0; n<nprocs; n++){
     printf("visione rank %d -> host %d %s\n",rank, n,host_names[n]);
   }
 }

 printf("rank %d -> colore %d\n",rank,color);

 MPI_Comm_split(MPI_COMM_WORLD, color, 0, &nodeComm);

 /// calcola rank sul singolo nodo (nodeComm)
 MPI_Comm_rank(nodeComm, &rank_node);
 MPI_Comm_size(nodeComm, &gpu_per_node);

 /* Find out how many DP capable GPUs are in the system and their device number */
 int deviceCount,slot=0;
 int *devloc;
 hipGetDeviceCount(&deviceCount);
 devloc=(int *)malloc(deviceCount*sizeof(int));
 devloc[0]=999;

 printf("Sono host %s, rank %d: vedo %d GPU, rank nel nodo %d\n",host_name,rank,deviceCount,rank_node);
 if (deviceCount<rank_node){
   printf("Warning: sul nodo sono previste meno GPU di rank!\n");
 }

 for (dev = 0; dev < deviceCount; ++dev)
   {
     hipGetDeviceProperties(&deviceProp, dev);
     if(deviceProp.major>1){
	 //printf("    --> rank %d (rank %d sul nodo): ct %d gpu %d\n",rank,rank_node,slot, dev);
	 devloc[slot]=dev;
	 slot++;
       };
   }
 // printf (" host %s rank nodo %d: Assigning device %d\n",	 host_name,rank_node, devloc[rank_node] );
 /* Assign device to MPI process and probe device properties */
 hipSetDevice(devloc[rank_node]);
 hipGetDevice(&dev);
 hipGetDeviceProperties(&deviceProp, dev);
 size_t free_bytes, total_bytes;
 hipMemGetInfo(&free_bytes, &total_bytes);
 printf("Host: %s Rank=%d RankNode=%d Device= %d (%s)  ECC=%s  Free = %lu, Total = %lu\n",host_name,rank, rank_node, devloc[rank_node],deviceProp.name, deviceProp.ECCEnabled ? "Enabled " : "Disabled", (unsigned long)free_bytes, (unsigned long)total_bytes);


 go(rank,rank_node);

 MPI_Finalize();
 }
