#include "hip/hip_runtime.h"
/*
heat_gpu.cu

nvcc -O2 heat_gpu.cu -o heat_gpu

./heat_gpu -h
./heat_gpu -r 4096 -c 4096
*/

#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <unistd.h>
#include <string.h>
#include <sys/time.h>  //gettimeofday

#define BLOCK_SIZE_X SIZE
#define BLOCK_SIZE_Y 16

__host__ __device__ int iDivUp(int a, int b){ return ((a % b) != 0) ? (a / b + 1) : (a / b); }

__global__ void Jacobi_Iterator_GPU(const float * __restrict__ T_old, float * __restrict__ T_new, const int NX, const int NY);
__global__ void copy_constant(float * __restrict__ T, const float * __restrict__ T_const, const int NX,  const int NY);

void options(int argc, char * argv[]) ;
void usage(char * argv[]);
void Jacobi_Iterator_CPU(float * __restrict T, float * __restrict T_new, const int NX, const int NY); 
void Init_center(float * __restrict h_T, const int NX, const int NY); // center 
void Init_left(float * __restrict h_T, const int NX, const int NY);   // left border
void Init_top(float * __restrict h_T, const int NX, const int NY);    // top border
void copy_rows(float * __restrict h_T, const int NX, const int NY);   // periodic boundary conditions
void copy_cols(float * __restrict h_T, const int NX, const int NY);   // periodic boundary conditions
void print_colormap(float * __restrict h_T);                          // 

int NX = 256;         // --- Number of discretization points along the x axis
int NY = 256;         // --- Number of discretization points along the y axis
int MAX_ITER = 1000;  // --- Number of Jacobi iterations

/********/
/* MAIN */
/********/
int main(int argc, char **argv)
{

  int iter;

  double t1, t2;
  struct timeval tempo;

  options(argc, argv);         /* optarg management */
  //fprintf (stderr,"# NX=%d, NY=%d, MAX_ITER=%d \n", NX, NY, MAX_ITER);

    // --- CPU temperature distributions
   float *h_T              = (float *)calloc(NX * NY, sizeof(float));
   Init_center(h_T,  NX, NY);
//   Init_left(h_T,    NX, NY);
    Init_top(h_T,     NX, NY);
   float *h_T_GPU_result   = (float *)malloc(NX * NY * sizeof(float));
   float *temp;

// --- GPU temperature distribution
    float *d_T;
    hipMalloc((void**)&d_T,      NX * NY * sizeof(float));
    float *d_T_old;
    hipMalloc((void**)&d_T_old,  NX * NY * sizeof(float));
    float *d_T_const;
    hipMalloc((void**)&d_T_const,  NX * NY * sizeof(float));

    hipMemcpy(d_T,     h_T, NX * NY * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_T_old, d_T, NX * NY * sizeof(float), hipMemcpyDeviceToDevice);
    hipMemcpy(d_T_const, d_T, NX * NY * sizeof(float), hipMemcpyDeviceToDevice);

    // --- Grid size
    dim3 dimBlock(BLOCK_SIZE_X, BLOCK_SIZE_Y);
    dim3 dimGrid (iDivUp(NX, BLOCK_SIZE_X), iDivUp(NY, BLOCK_SIZE_Y));

///////////////////////////////


   gettimeofday(&tempo,0);  t1=tempo.tv_sec+(tempo.tv_usec/1000000.0); // get timer1

   for(iter=0; iter<MAX_ITER; iter=iter+1)
    {

    copy_constant<<<dimGrid, dimBlock>>>(d_T, d_T_const, NX, NY);
    Jacobi_Iterator_GPU<<<dimGrid, dimBlock>>>(d_T, d_T_old, NX, NY); 

    temp=d_T;
    d_T=d_T_old;
    d_T_old=temp;
    }

    gettimeofday(&tempo,0);  t2=tempo.tv_sec+(tempo.tv_usec/1000000.0); // get timer2

    // --- Copy result from device to host
    hipMemcpy(h_T_GPU_result, d_T, NX * NY * sizeof(float), hipMemcpyDeviceToHost);

    fprintf (stderr,"%.3f \n", t2-t1);
    //print_colormap(h_T_GPU_result);

    free(h_T);
    hipFree(d_T);
    hipFree(d_T_old);

    return 0;
}


/***********************************/
/* JACOBI ITERATION FUNCTION - CPU */
/***********************************/
void Jacobi_Iterator_CPU(float * __restrict T, float * __restrict T_new, const int NX, const int NY)
{
int i,j;

        // --- Only update "interior" (not boundary) node points
        for(j=1; j<NY-1; j++)
            for(i=1; i<NX-1; i++) {
                float T_E = T[(i+1) + NX*j];
                float T_W = T[(i-1) + NX*j];
                float T_N = T[i + NX*(j+1)];
                float T_S = T[i + NX*(j-1)];
                T_new[NX*j + i] = 0.25*(T_E + T_W + T_N + T_S);
            }
}

/*************
***** copy constant
************/

__global__ void copy_constant(float * __restrict__ T, const float * __restrict__ T_const, const int NX, const int NY)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x ;
    const int j = blockIdx.y * blockDim.y + threadIdx.y ;

    int P = i + j*NX;
    if(T_const[P] > 0) T[P] = T_const[P];              // copia punti a temperatura costante

    if (j==0)           T[NX*0+i]=     T[NX*(NY-2)+i]; // copia penultima riga nella prima
    if (j==(NY-1))      T[NX*(NY-1)+i]=T[(NX*1)+i];    // copia seconda riga nell'ultima riga
    if (i==0)           T[NX*j+0]=     T[NX*j+NX-2];   // copia penultima colonna nella prima
    if (i==(NX-1))      T[NX*j+(NX-1)]=T[(NX*j)+1];    // copia seconda colonna nell'ultima

}


/***********************************/
/* JACOBI ITERATION FUNCTION - GPU */
/***********************************/
__global__ void Jacobi_Iterator_GPU(const float * __restrict__ T_old, float * __restrict__ T_new, const int NX, const int NY)
{
    __shared__ float T_shared[(BLOCK_SIZE_Y+2)*(BLOCK_SIZE_X +2)];

    // indici per la matrice globale T_old 
    const int global_j = blockIdx.x * blockDim.x + threadIdx.x ;
    const int global_i = blockIdx.y * blockDim.y + threadIdx.y ;

    //indici per il riempimento della matrice shared
    const int shared_j = threadIdx.x + 1;
    const int shared_i = threadIdx.y + 1;

    const int size = blockDim.x+2;

    //Riempimento della mia matrice shared
    // Se incontro le celle di bordo allora l'esterno della mia matrice shared lo imposto uguale a 0 così non influisce sul risultato finale
    // Se invece la cella di un blocco è sul bordo del suo blocco ma interna sulla matrice globale allora faccio scrivere il suo bordo con la relativa cella (sopra/sotto/destra/sinistra)
    if(shared_i == 1){
        if(global_i > 0)
            T_shared[(shared_i-1)*size + shared_j] = T_old[(global_i-1)*NX + global_j];
        else
            T_shared[(shared_i-1)*size + shared_j] = 0;
    }
    else if(shared_i == blockDim.y){
        if(global_i < NY-1)
            T_shared[(shared_i+1)* size + shared_j] = T_old[(global_i+1)*NX + global_j];
        else
            T_shared[(shared_i+1)* size + shared_j] = 0;

    }
    if(shared_j == 1){
        if(global_j > 0)
            T_shared[shared_i*size + shared_j-1] = T_old[global_i*NX + global_j-1];
        else
            T_shared[shared_i*size + shared_j-1] = 0;
    }
    else if(shared_j == blockDim.x){
        if(global_j < NX-1)
            T_shared[shared_i*size + shared_j+1] = T_old[global_i*NX + global_j+1];
        else
            T_shared[shared_i*size + shared_j+1] = 0;
    }
    
    T_shared[shared_i *size + shared_j] = T_old[global_i*NX + global_j];

    __syncthreads();

    int left=shared_j-1;
    int right=shared_j+1;
    int up=shared_i-1;
    int down=shared_i+1;
    /*
    if (left<0) left=0;
    if (right>=gridDim.x*blockDim.x) right--;
    if (down<0) down=0;
    if (up>=gridDim.y*blockDim.y) up--;
    */
                              //                                                N
    int global_P = global_j + global_i*NX;         // node (i,j)                |
    int N = shared_j + up*size;        // node (i,j+1)                          |
    int S = shared_j + down*size;      // node (i,j-1)                   W ---- P ---- E
    int E = right + shared_i*size;     // node (i+1,j)                          |
    int W = left + shared_i*size;      // node (i-1,j)                          |
                              //                                                S
    /// Update
    T_new[global_P] = 0.25 * (T_shared[E] + T_shared[W] + T_shared[N] + T_shared[S]);
}


/********************************/
/* TEMPERATURE INITIALIZATION : */
/* parte centrale della griglia */
/********************************/
void Init_center(float * __restrict h_T, const int NX, const int NY)
{
    int i,j;
    int startx=NX/2-NX/10;
    int endx=NX/2+NX/10;
    int starty=NY/2-NY/10;
    int endy=NY/2+NY/10;
//    int starty=NY/4;
//    int endy=NY-NY/4;
    for(i=startx; i<endx; i++)
        for(j=starty; j<endy; j++)
              h_T[NX*j + i] = 1.0;
}


/********************************/
/* TEMPERATURE INITIALIZATION : */
/* bordo sinistro               */
/********************************/
void Init_left(float * __restrict h_T, const int NX, const int NY)
{

    int i,j;
    int startx=1;
    int endx=2;
    int starty=0;
    int endy=NY-1;
    for(i=startx; i<endx; i++)
        for(j=starty; j<endy; j++)
              h_T[NX*j + i] = 1.0;
}


/********************************/
/* TEMPERATURE INITIALIZATION : */
/* bordo alto                   */
/********************************/
void Init_top(float * __restrict h_T, const int NX, const int NY)
{

    int i;
    int startx=0;
    int endx=NX-1;
    for(i=startx; i<endx; i++)
              h_T[NX + i] = 1.0;
}


/********************************/
/* Periodic boundary conditions */
/* COPY BORDER: COLS            */
/********************************/
void copy_cols (float * __restrict h_T, const int NX, const int NY)
{

int i;

// copy cols
  for (i = 1; i < NY-1; ++i) {
    h_T[NX*i+0]    = h_T[NX*i+NX-2];
    h_T[NX*i+NX-1] = h_T[NX*i+1];
  }
}



/********************************/
/* Periodic boundary conditions */
/* COPY BOREDER: ROWS           */
/********************************/
void copy_rows (float * __restrict h_T, const int NX, const int NY)
{

   memcpy(&(h_T[NX*0])      ,&(h_T[NX*(NY-2)]), NX*sizeof(float) );
   memcpy(&(h_T[NX*(NY-1)]) ,&(h_T[NX*1]),      NX*sizeof(float) );
}




/******************************************/
/* print color map                        */
/******************************************/

void print_colormap(float * __restrict h_T)
{
   int i,j;

   for (j=1; j<NY-1; j++){
        for (i=1; i<NX-1; i++) {
            printf("%2.2f ",h_T[NX*j + i]);
        }
        printf("\n");
        }
}


/******************************************/
/* options management                     */
/******************************************/
void options(int argc, char * argv[]) {

  int i;
   while ( (i = getopt(argc, argv, "c:r:s:h")) != -1) {
        switch (i) {
        case 'c':  NX       = strtol(optarg, NULL, 10);  break;
        case 'r':  NY       = strtol(optarg, NULL, 10);  break;
        case 's':  MAX_ITER = strtol(optarg, NULL, 10);  break;
        case 'h':  usage(argv); exit(1);
        case '?':  usage(argv); exit(1);
        default:   usage(argv); exit(1);
        }
    }
}

/******************************************/
/* print help                             */
/******************************************/
void usage(char * argv[])  {

  printf ("\n%s [-c ncols] [-r nrows] [-s nsteps] [-h]",argv[0]);
  printf ("\n");

}

