/*
heat_gpu.cu

nvcc -O2 heat_gpu.cu -o heat_gpu

./heat_gpu -h
./heat_gpu -r 4096 -c 4096
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <unistd.h>
#include <string.h>
#include <sys/time.h>  //gettimeofday

#define BLOCK_SIZE_X 16
#define BLOCK_SIZE_Y 16

__host__ __device__ int iDivUp(int a, int b){ return ((a % b) != 0) ? (a / b + 1) : (a / b); }

__global__ void Jacobi_Iterator_GPU(const float * __restrict__ T_old, float * __restrict__ T_new, const int NX, const int NY);
__global__ void copy_constant(float * __restrict__ T, const float * __restrict__ T_const, const int NX,  const int NY);

void options(int argc, char * argv[]) ;
void usage(char * argv[]);
void Jacobi_Iterator_CPU(float * __restrict T, float * __restrict T_new, const int NX, const int NY); 
void Init_center(float * __restrict h_T, const int NX, const int NY); // center 
void Init_left(float * __restrict h_T, const int NX, const int NY);   // left border
void Init_top(float * __restrict h_T, const int NX, const int NY);    // top border
void copy_rows(float * __restrict h_T, const int NX, const int NY);   // periodic boundary conditions
void copy_cols(float * __restrict h_T, const int NX, const int NY);   // periodic boundary conditions
void print_colormap(float * __restrict h_T);                          // 

int NX = 256;         // --- Number of discretization points along the x axis
int NY = 256;         // --- Number of discretization points along the y axis
int MAX_ITER = 1000;  // --- Number of Jacobi iterations

/********/
/* MAIN */
/********/
int main(int argc, char **argv)
{

  int iter;

  double t1, t2;
  struct timeval tempo;

  options(argc, argv);         /* optarg management */
  //fprintf (stderr,"# NX=%d, NY=%d, MAX_ITER=%d \n", NX, NY, MAX_ITER);

    // --- CPU temperature distributions
   float *h_T              = (float *)calloc(NX * NY, sizeof(float));
   Init_center(h_T,  NX, NY);
//   Init_left(h_T,    NX, NY);
    Init_top(h_T,     NX, NY);
   float *h_T_GPU_result   = (float *)malloc(NX * NY * sizeof(float));
   float *temp;

// --- GPU temperature distribution
    float *d_T;
    hipMalloc((void**)&d_T,      NX * NY * sizeof(float));
    float *d_T_old;
    hipMalloc((void**)&d_T_old,  NX * NY * sizeof(float));
    float *d_T_const;
    hipMalloc((void**)&d_T_const,  NX * NY * sizeof(float));

    hipMemcpy(d_T,     h_T, NX * NY * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_T_old, d_T, NX * NY * sizeof(float), hipMemcpyDeviceToDevice);
    hipMemcpy(d_T_const, d_T, NX * NY * sizeof(float), hipMemcpyDeviceToDevice);

    // --- Grid size
    dim3 dimBlock(BLOCK_SIZE_X, BLOCK_SIZE_Y);
    dim3 dimGrid (iDivUp(NX, BLOCK_SIZE_X), iDivUp(NY, BLOCK_SIZE_Y));

///////////////////////////////


   gettimeofday(&tempo,0);  t1=tempo.tv_sec+(tempo.tv_usec/1000000.0); // get timer1

   for(iter=0; iter<MAX_ITER; iter=iter+1)
    {

    copy_constant<<<dimGrid, dimBlock>>>(d_T, d_T_const, NX, NY);
    Jacobi_Iterator_GPU<<<dimGrid, dimBlock>>>(d_T, d_T_old, NX, NY); 

    temp=d_T;
    d_T=d_T_old;
    d_T_old=temp;
    }

    gettimeofday(&tempo,0);  t2=tempo.tv_sec+(tempo.tv_usec/1000000.0); // get timer2

    // --- Copy result from device to host
    hipMemcpy(h_T_GPU_result, d_T, NX * NY * sizeof(float), hipMemcpyDeviceToHost);

    fprintf (stderr,"%.3f\n", t2-t1);
    //print_colormap(h_T_GPU_result);

    free(h_T);
    hipFree(d_T);
    hipFree(d_T_old);

    return 0;
}


/***********************************/
/* JACOBI ITERATION FUNCTION - CPU */
/***********************************/
void Jacobi_Iterator_CPU(float * __restrict T, float * __restrict T_new, const int NX, const int NY)
{
int i,j;

        // --- Only update "interior" (not boundary) node points
        for(j=1; j<NY-1; j++)
            for(i=1; i<NX-1; i++) {
                float T_E = T[(i+1) + NX*j];
                float T_W = T[(i-1) + NX*j];
                float T_N = T[i + NX*(j+1)];
                float T_S = T[i + NX*(j-1)];
                T_new[NX*j + i] = 0.25*(T_E + T_W + T_N + T_S);
            }
}

/*************
***** copy constant
************/

__global__ void copy_constant(float * __restrict__ T, const float * __restrict__ T_const, const int NX, const int NY)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x ;
    const int j = blockIdx.y * blockDim.y + threadIdx.y ;

    int P = i + j*NX;
    if(T_const[P] > 0) T[P] = T_const[P];              // copia punti a temperatura costante

    if (j==0)           T[NX*0+i]=     T[NX*(NY-2)+i]; // copia penultima riga nella prima
    if (j==(NY-1))      T[NX*(NY-1)+i]=T[(NX*1)+i];    // copia seconda riga nell'ultima riga
    if (i==0)           T[NX*j+0]=     T[NX*j+NX-2];   // copia penultima colonna nella prima
    if (i==(NX-1))      T[NX*j+(NX-1)]=T[(NX*j)+1];    // copia seconda colonna nell'ultima

}


/***********************************/
/* JACOBI ITERATION FUNCTION - GPU */
/***********************************/
__global__ void Jacobi_Iterator_GPU(const float * __restrict__ T_old, float * __restrict__ T_new, const int NX, const int NY)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x ;
    const int j = blockIdx.y * blockDim.y + threadIdx.y ;

    int left=i-1;
    int right=i+1;
    int up=j+1;
    int down=j-1;
    if (left<0) left=0;
    if (right>=gridDim.x*blockDim.x) right--;
    if (down<0) down=0;
    if (up>=gridDim.y*blockDim.y) up--;
                              //                         N
    int P = i + j*NX;         // node (i,j)              |
    int N = i + up*NX;        // node (i,j+1)            |
    int S = i + down*NX;      // node (i,j-1)     W ---- P ---- E
    int E = right + j*NX;     // node (i+1,j)            |
    int W = left + j*NX;      // node (i-1,j)            |
                              //                         S
    /// Update
    T_new[P] = 0.25 * (T_old[E] + T_old[W] + T_old[N] + T_old[S]);
}


/********************************/
/* TEMPERATURE INITIALIZATION : */
/* parte centrale della griglia */
/********************************/
void Init_center(float * __restrict h_T, const int NX, const int NY)
{
    int i,j;
    int startx=NX/2-NX/10;
    int endx=NX/2+NX/10;
    int starty=NY/2-NY/10;
    int endy=NY/2+NY/10;
//    int starty=NY/4;
//    int endy=NY-NY/4;
    for(i=startx; i<endx; i++)
        for(j=starty; j<endy; j++)
              h_T[NX*j + i] = 1.0;
}


/********************************/
/* TEMPERATURE INITIALIZATION : */
/* bordo sinistro               */
/********************************/
void Init_left(float * __restrict h_T, const int NX, const int NY)
{

    int i,j;
    int startx=1;
    int endx=2;
    int starty=0;
    int endy=NY-1;
    for(i=startx; i<endx; i++)
        for(j=starty; j<endy; j++)
              h_T[NX*j + i] = 1.0;
}


/********************************/
/* TEMPERATURE INITIALIZATION : */
/* bordo alto                   */
/********************************/
void Init_top(float * __restrict h_T, const int NX, const int NY)
{

    int i;
    int startx=0;
    int endx=NX-1;
    for(i=startx; i<endx; i++)
              h_T[NX + i] = 1.0;
}


/********************************/
/* Periodic boundary conditions */
/* COPY BORDER: COLS            */
/********************************/
void copy_cols (float * __restrict h_T, const int NX, const int NY)
{

int i;

// copy cols
  for (i = 1; i < NY-1; ++i) {
    h_T[NX*i+0]    = h_T[NX*i+NX-2];
    h_T[NX*i+NX-1] = h_T[NX*i+1];
  }
}



/********************************/
/* Periodic boundary conditions */
/* COPY BOREDER: ROWS           */
/********************************/
void copy_rows (float * __restrict h_T, const int NX, const int NY)
{

   memcpy(&(h_T[NX*0])      ,&(h_T[NX*(NY-2)]), NX*sizeof(float) );
   memcpy(&(h_T[NX*(NY-1)]) ,&(h_T[NX*1]),      NX*sizeof(float) );
}




/******************************************/
/* print color map                        */
/******************************************/

void print_colormap(float * __restrict h_T)
{
   int i,j;

   for (j=1; j<NY-1; j++){
        for (i=1; i<NX-1; i++) {
            printf("%2.2f ",h_T[NX*j + i]);
        }
        printf("\n");
        }
}


/******************************************/
/* options management                     */
/******************************************/
void options(int argc, char * argv[]) {

  int i;
   while ( (i = getopt(argc, argv, "c:r:s:h")) != -1) {
        switch (i) {
        case 'c':  NX       = strtol(optarg, NULL, 10);  break;
        case 'r':  NY       = strtol(optarg, NULL, 10);  break;
        case 's':  MAX_ITER = strtol(optarg, NULL, 10);  break;
        case 'h':  usage(argv); exit(1);
        case '?':  usage(argv); exit(1);
        default:   usage(argv); exit(1);
        }
    }
}

/******************************************/
/* print help                             */
/******************************************/
void usage(char * argv[])  {

  printf ("\n%s [-c ncols] [-r nrows] [-s nsteps] [-h]",argv[0]);
  printf ("\n");

}

